#include "hip/hip_runtime.h"
#include "pp_dynamic_access_offchip_memory_vecadd_repeat.h"

__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
  int blockStartIndex  = blockIdx.x * BLOCK_WIDTH * VALUES_PER_THREAD;
  int threadStartIndex = blockStartIndex;
  //int threadStartIndex = blockStartIndex + threadIdx.x;
  int threadEndIndex   = threadStartIndex + VALUES_PER_THREAD*BLOCK_WIDTH;
  //int threadEndIndex   = threadStartIndex + VALUES_PER_THREAD*blockDim.x;
  int i,t,s;
 
  __shared__ float shared_C[SHARED_ARRAY_SIZE]; 

  for (t = 0; t < REPS; t++) {
    //BLOCK_WIDTH*VALUES_PER_THREAD should be divicible by SHARED_ARRAY_SIZE
    for (s = threadStartIndex; s < threadEndIndex; s=s+SHARED_ARRAY_SIZE) {
      //SHARED_ARRAY_SIZE should be divicible by BLOCK_WIDTH
      //for( i=s ; i<s+SHARED_ARRAY_SIZE; i=i+BLOCK_WIDTH ){
      for( i=s+threadIdx.x ; i<s+SHARED_ARRAY_SIZE; i=i+BLOCK_WIDTH ){
          shared_C[i-s] = A[i] + B[i];
      }
      __syncthreads();

      for( i=s+threadIdx.x ; i<s+SHARED_ARRAY_SIZE; i=i+BLOCK_WIDTH ){
          C[i] = shared_C[i-s];
      }
    }
  }
}
