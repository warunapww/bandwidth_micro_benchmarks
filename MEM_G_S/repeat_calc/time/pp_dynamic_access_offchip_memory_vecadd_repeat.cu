#include "hip/hip_runtime.h"
///
/// vecadd.cu
/// For CSU CS575 Spring 2011
/// Instructor: Wim Bohm
/// Based on code from the CUDA Programming Guide
/// Modified by Wim Bohm and David Newman
/// Created: 2011-02-03
/// Last Modified: 2011-03-03 DVN
///
/// Add two Vectors A and B in C on GPU using
/// a kernel defined according to vecAddKernel.h
/// Students must not modify this file. The GTA
/// will grade your submission using an unmodified
/// copy of this file.
/// 

// Includes
#include <stdio.h>
#include "pp_dynamic_access_offchip_memory_vecadd_repeat.h"

#include "high_resolution_power.h"

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {                    \
    hipError_t _m_cudaStat = value;                    \
    if (_m_cudaStat != hipSuccess) {                   \
      fprintf(stderr, "Error: %s at line %d in file %s\n",        \
          hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);   \
          exit(1);        \
    } }




// Variables for host and device vectors.
float* h_A; 
float* h_B; 
float* h_C; 
float* d_A; 
float* d_B; 
float* d_C; 

// Utility Functions
void Cleanup(bool);

void call_gpu_function() {
    dim3 dimGrid(GRID_WIDTH);                    
    dim3 dimBlock(BLOCK_WIDTH);                 
    AddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, VALUES_PER_THREAD);
}

// Host code performs setup and calls the kernel.
int main(int argc, char** argv)
{
    int N; //Vector size

	// Parse arguments.
/*    if(argc != 2){
     printf("Usage: %s VALUES_PER_THREAD\n", argv[0]);
     printf("VALUES_PER_THREAD is the number of values added by each thread.\n");
     printf("Total vector size is 128 * 60 * this value.\n");
     exit(0);
    } else {
      sscanf(argv[1], "%d", &VALUES_PER_THREAD);
    }      
*/
    // Determine the number of threads .
    // N is the total number of values to be in a vector
    N = VALUES_PER_THREAD * GRID_WIDTH * BLOCK_WIDTH;
    printf("Total vector size: %d\n", N); 
    // size_t is the total number of bytes for a vector.
    size_t size = N * sizeof(float);

    // Tell CUDA how big to make the grid and thread blocks.
    // Since this is a vector addition problem,
    // grid and thread block are both one-dimensional.

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) Cleanup(false);
    h_B = (float*)malloc(size);
    if (h_B == 0) Cleanup(false);
    h_C = (float*)malloc(size);
    if (h_C == 0) Cleanup(false);

    // Allocate vectors in device memory.
    hipError_t error;
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_A, size));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_B, size));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_C, size));

    // Initialize host vectors h_A and h_B
    int i;
    for(i=0; i<N; ++i){
     h_A[i] = (float)i;
     h_B[i] = (float)(N-i);   
     h_C[i] = (float)0.0;
    }

    // Copy host vectors h_A and h_B to device vectores d_A and d_B
    CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice));
	  CUDA_CHECK_RETURN(hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice));

    // Warm up
//    AddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, VALUES_PER_THREAD);
    call_gpu_function();
    CUDA_CHECK_RETURN(hipGetLastError());
    CUDA_CHECK_RETURN(hipDeviceSynchronize());


    long long exec_time_nanoseconds = get_exec_time_in_nanoseconds(call_gpu_function); 
    
    //high_resolution_power_profile(call_gpu_function);


    double time = exec_time_nanoseconds/1e6; //in ms

    //high_resolution_power_profile(call_gpu_function);


	// Compute floating point operations per second.
    double nFlops = (double)N*(double)REPS ;
    double nFlopsPerSec = 1e3*nFlops/time;
    double nGFlopsPerSec = nFlopsPerSec*1e-9;
    //double nGFlopsPerSec = (1e3*N/(exec_time_nanoseconds/1e9))*1e-9;
    //double nGFlopsPerSec = 1e3*N/exec_time_nanoseconds;

	// Compute transfer rates.
    double nBytes = 3*4*(double)N*(double)REPS; // 2N words in, 1N word out
    double nBytesPerSec = 1e3*nBytes/time;
    double nGBytesPerSec = nBytesPerSec*1e-9;
    //double nGBytesPerSec = (1e3*nBytes/(exec_time_nanoseconds/1e9))*1e-9;
    //double nGBytesPerSec = 1e3*nBytes/exec_time_nanoseconds;

	// Report timing data.
    printf( "%d %d %d Time: %f (ms), GFlopsS: %f GBytesS: %f nytes: %f nBytesPerS: %f\n", GRID_WIDTH, BLOCK_WIDTH, VALUES_PER_THREAD,
             time, nGFlopsPerSec, nGBytesPerSec, nBytes, nBytesPerSec);
     
    // Copy result from device memory to host memory
    error = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) Cleanup(false);

    // Verify & report result
    for (i = 0; i < N; ++i) {
        float val = h_C[i];
        if (fabs(val - N) > 1e-5)
            break;
    }
    printf("Test %s \n", (i == N) ? "PASSED" : "FAILED");

	// Clean up and exit.
    Cleanup(true);
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
        
    // Free device vectors
    if (d_A)
        CUDA_CHECK_RETURN(hipFree(d_A));
    if (d_B)
        CUDA_CHECK_RETURN(hipFree(d_B));
    if (d_C)
        CUDA_CHECK_RETURN(hipFree(d_C));

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    error = hipDeviceReset();
    
    if (!noError || error != hipSuccess)
        printf("cuda malloc or cuda thread exit failed \n");
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}



