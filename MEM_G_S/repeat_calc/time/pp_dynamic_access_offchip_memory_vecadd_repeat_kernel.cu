#include "hip/hip_runtime.h"
/*
This kernel does the vector addition and repeat the same addition for REP times.

The computations are repeated inorder to have a considerable execution time (few seconds).
The aim is to find the energy to transfer data between registers and shared memory

Egs - Energy to transfer an element between global memory and shared memory
Egr - Energy to transfer an element between global memory and registers
Esr - Energy to transfer an element between registers and shared memory

Egs = Egr + Esr

Dynanic energy = Egs*(#global-shared transfers) + Esr*(#shared-register transfers)

Known params:
  Egr from preveous experiment
  #global-shared transfers
  #shared-register transfers
  Dynanic energy - Energy for the program - static energy
Unknown params:
  Esr (Egs can be represented using Egr and Esr using the above equation)

*/
#include "pp_dynamic_access_offchip_memory_vecadd_repeat.h"

__global__ void AddVectors(const float* A, const float* B, float* C, int N)
{
  int blockStartIndex  = blockIdx.x * BLOCK_WIDTH * VALUES_PER_THREAD;
  int threadStartIndex = blockStartIndex;
  //int threadStartIndex = blockStartIndex + threadIdx.x;
  int threadEndIndex   = threadStartIndex + VALUES_PER_THREAD*BLOCK_WIDTH;
  //int threadEndIndex   = threadStartIndex + VALUES_PER_THREAD*blockDim.x;
  int i,t,s;
 
  __shared__ float shared_C[SHARED_ARRAY_SIZE]; 

  for (t = 0; t < REPS; t++) {
    //BLOCK_WIDTH*VALUES_PER_THREAD should be divicible by SHARED_ARRAY_SIZE
    for (s = threadStartIndex; s < threadEndIndex; s=s+SHARED_ARRAY_SIZE) {
      //SHARED_ARRAY_SIZE should be divicible by BLOCK_WIDTH
      for( i=s+threadIdx.x ; i<s+SHARED_ARRAY_SIZE; i=i+BLOCK_WIDTH ){
          shared_C[i-s] = A[i] + B[i];
      }
      __syncthreads();

      for( i=s+threadIdx.x ; i<s+SHARED_ARRAY_SIZE; i=i+BLOCK_WIDTH ){
          C[i] = shared_C[i-s];
      }
    }
  }
}
