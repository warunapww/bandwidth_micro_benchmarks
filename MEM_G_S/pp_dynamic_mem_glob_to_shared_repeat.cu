#include "hip/hip_runtime.h"
///
/// vecadd.cu
/// For CSU CS575 Spring 2011
/// Instructor: Wim Bohm
/// Based on code from the CUDA Programming Guide
/// Modified by Wim Bohm and David Newman
/// Created: 2011-02-03
/// Last Modified: 2011-03-03 DVN
///
/// Add two Vectors A and B in C on GPU using
/// a kernel defined according to vecAddKernel.h
/// Students must not modify this file. The GTA
/// will grade your submission using an unmodified
/// copy of this file.
/// 

// Includes
#include <stdio.h>
#include "pp_dynamic_mem_glob_to_shared_repeat.h"

#include "high_resolution_power.h"

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {                    \
    hipError_t _m_cudaStat = value;                    \
    if (_m_cudaStat != hipSuccess) {                   \
      fprintf(stderr, "Error: %s at line %d in file %s\n",        \
          hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);   \
          exit(1);        \
    } }


// Variables for host and device vectors.
float* h_A; 
float* h_B; 
float* d_A; 
float* d_B; 
int ValuesPerThread; // number of values per thread
int N; //Vector size

// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);

void call_gpu_function() {
    dim3 dimGrid(GRID_WIDTH);                    
    dim3 dimBlock(BLOCK_WIDTH);                 
    reverse_in_chunks<<<dimGrid, dimBlock>>>(d_A, d_B, ValuesPerThread, N);
}

// Host code performs setup and calls the kernel.
int main(int argc, char** argv)
{

	// Parse arguments.
    if(argc != 2){
     printf("Usage: %s ValuesPerThread\n", argv[0]);
     printf("ValuesPerThread is the number of values added by each thread.\n");
     printf("Total vector size is 128 * 60 * this value.\n");
     exit(0);
    } else {
      sscanf(argv[1], "%d", &ValuesPerThread);
    }      

    // Determine the number of threads .
    // N is the total number of values to be in a vector
    N = ValuesPerThread * GRID_WIDTH * BLOCK_WIDTH;
    printf("Total vector size: %d\n", N); 
    // size_t is the total number of bytes for a vector.
    size_t size = N * sizeof(float);

    // Tell CUDA how big to make the grid and thread blocks.
    // Since this is a vector addition problem,
    // grid and thread block are both one-dimensional.

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) Cleanup(false);
    h_B = (float*)malloc(size);
    if (h_B == 0) Cleanup(false);

    // Allocate vectors in device memory.
    hipError_t error;
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_A, size));
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_B, size));

    // Initialize host vectors h_A and h_B
    int i;
    for(i=0; i<N; ++i){
     h_A[i] = (float)i;
    }

    // Copy host vectors h_A and h_B to device vectores d_A and d_B
    CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

    // Warm up
//    AddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread);
    call_gpu_function();
    CUDA_CHECK_RETURN(hipGetLastError());
    CUDA_CHECK_RETURN(hipDeviceSynchronize());


    // Invoke kernel
//    AddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread);
    call_gpu_function();
    CUDA_CHECK_RETURN(hipGetLastError());

	// Compute elapsed time 
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    long long exec_time_nanoseconds = get_exec_time_in_nanoseconds(call_gpu_function); 
    
    //high_resolution_power_profile(call_gpu_function);


    double time = exec_time_nanoseconds/1e6; //in ms

	// Compute floating point operations per second.
/*    double nFlops = (double)N*(double)REPS ;
    double nFlopsPerSec = 1e3*nFlops/time;
    double nGFlopsPerSec = nFlopsPerSec*1e-9;
    //double nGFlopsPerSec = (1e3*N/(exec_time_nanoseconds/1e9))*1e-9;
    //double nGFlopsPerSec = 1e3*N/exec_time_nanoseconds;
*/
	// Compute transfer rates.
    double nBytes =2*4*(double)N*(double)REPS; // N words in, N word out
    double nBytesPerSec = 1e3*nBytes/time;
    double nGBytesPerSec = nBytesPerSec*1e-9;
    //double nGBytesPerSec = (1e3*nBytes/(exec_time_nanoseconds/1e9))*1e-9;
    //double nGBytesPerSec = 1e3*nBytes/exec_time_nanoseconds;

	// Report timing data.
    printf( "GRID_WIDTH: %d BLOCK_WIDTH: %d ValuesPerThread: %d REPS: %d Time: %f (ms), GBytesS: %f\n", GRID_WIDTH, BLOCK_WIDTH, ValuesPerThread, REPS,
             time, nGBytesPerSec);
     
    // Copy result from device memory to host memory
    error = hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) Cleanup(false);

    // Verify & report result
   /* //for (t = 0; t < REPS; t++) {
      for (j = 0; j < times; j++) {
        int threadStartIndex = blockStartIndex + 12288*j;
        int threadEndIndex   = threadStartIndex + 12288;
        for( i=threadStartIndex; i<threadEndIndex; i=i+BLOCK_WIDTH){
          int k = i+tId;
          //sharedA[tId] = A[k];
          sharedA[k-threadStartIndex] = A[k];
        }
        __syncthreads();

        for( i=threadStartIndex; i<threadEndIndex; i=i+BLOCK_WIDTH){
          int k = i+tId;
          B[L-k-12288] = sharedA[k-threadStartIndex];
        }

      
      }
    //}

    for (i = 0; i < N; ++i) {
        float val = h_A[i];
        if (fabs(val - h_B[N-(i/12288)*12288 + i%12288]) > 1e-5)
            break;
    }
    printf("Test %s \n", (i == N) ? "PASSED" : "FAILED");
*/
	// Clean up and exit.
    Cleanup(true);
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
        
    // Free device vectors
    if (d_A) {
        CUDA_CHECK_RETURN(hipFree(d_A));
    }
    if (d_B) {
        CUDA_CHECK_RETURN(hipFree(d_B));
    }

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
        
    error = hipDeviceReset();
    
    if (!noError || error != hipSuccess)
        printf("cuda malloc or cuda thread exit failed \n");
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}



