#include "hip/hip_runtime.h"
#include "pp_dynamic_mem_glob_to_shared_repeat.h"

__global__ void reverse_in_chunks(const float* A, float* B, int N, int L)
{
  int blockStartIndex  = blockIdx.x * BLOCK_WIDTH * N;
  int i,j,t;

  int tId = threadIdx.x; 
  int times = BLOCK_WIDTH*N/SHARED_ARRAY_SIZE;


  __shared__ float sharedA[SHARED_ARRAY_SIZE];
  //__shared__ float sharedA[SHARED_ARRAY_SIZE_WO_BANK_CONFLICTS];


  for (t = 0; t < REPS; t++) {
    for (j = 0; j < times; j++) {
      int threadStartIndex = blockStartIndex + SHARED_ARRAY_SIZE*j;
      int threadEndIndex   = threadStartIndex + SHARED_ARRAY_SIZE;
      for( i=threadStartIndex; i<threadEndIndex; i=i+BLOCK_WIDTH){
          int k = i+tId;
          int shared_k = k-threadStartIndex;
          //int shared_k = k-threadStartIndex + ((k-threadStartIndex)>>5);
          //sharedA[tId] = A[k];
          sharedA[shared_k] = A[k];
      }
      __syncthreads();

      for( i=threadStartIndex; i<threadEndIndex; i=i+BLOCK_WIDTH){
          int k = i+tId;
          int shared_k = k-threadStartIndex;
          //int shared_k = k-threadStartIndex + ((k-threadStartIndex)>>5);
          B[L-k-SHARED_ARRAY_SIZE] = sharedA[shared_k];
      }

      
    }
  }
}
