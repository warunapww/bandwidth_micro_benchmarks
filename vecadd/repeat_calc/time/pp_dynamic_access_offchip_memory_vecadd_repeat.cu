#include "hip/hip_runtime.h"
///
/// vecadd.cu
/// For CSU CS575 Spring 2011
/// Instructor: Wim Bohm
/// Based on code from the CUDA Programming Guide
/// Modified by Wim Bohm and David Newman
/// Created: 2011-02-03
/// Last Modified: 2011-03-03 DVN
///
/// Add two Vectors A and B in C on GPU using
/// a kernel defined according to vecAddKernel.h
/// Students must not modify this file. The GTA
/// will grade your submission using an unmodified
/// copy of this file.
/// 

// Includes
#include <stdio.h>
#include "pp_dynamic_access_offchip_memory_vecadd_repeat.h"

#include "high_resolution_power.h"


// Variables for host and device vectors.
float* h_A; 
float* h_B; 
float* h_C; 
float* d_A; 
float* d_B; 
float* d_C; 
int ValuesPerThread; // number of values per thread

// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);

void call_gpu_function() {
    dim3 dimGrid(GridWidth);                    
    dim3 dimBlock(BlockWidth);                 
    AddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread);
}

// Host code performs setup and calls the kernel.
int main(int argc, char** argv)
{
    int N; //Vector size

	// Parse arguments.
    if(argc != 2){
     printf("Usage: %s ValuesPerThread\n", argv[0]);
     printf("ValuesPerThread is the number of values added by each thread.\n");
     printf("Total vector size is 128 * 60 * this value.\n");
     exit(0);
    } else {
      sscanf(argv[1], "%d", &ValuesPerThread);
    }      

    // Determine the number of threads .
    // N is the total number of values to be in a vector
    N = ValuesPerThread * GridWidth * BlockWidth;
    printf("Total vector size: %d\n", N); 
    // size_t is the total number of bytes for a vector.
    size_t size = N * sizeof(float);

    // Tell CUDA how big to make the grid and thread blocks.
    // Since this is a vector addition problem,
    // grid and thread block are both one-dimensional.

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    if (h_A == 0) Cleanup(false);
    h_B = (float*)malloc(size);
    if (h_B == 0) Cleanup(false);
    h_C = (float*)malloc(size);
    if (h_C == 0) Cleanup(false);

    // Allocate vectors in device memory.
    hipError_t error;
    error = hipMalloc((void**)&d_A, size);
    if (error != hipSuccess) Cleanup(false);
    error = hipMalloc((void**)&d_B, size);
    if (error != hipSuccess) Cleanup(false);
    error = hipMalloc((void**)&d_C, size);
    if (error != hipSuccess) Cleanup(false);

    // Initialize host vectors h_A and h_B
    int i;
    for(i=0; i<N; ++i){
     h_A[i] = (float)i;
     h_B[i] = (float)(N-i);   
	 h_C[i] = (float)0.0;
    }

    // Copy host vectors h_A and h_B to device vectores d_A and d_B
    error = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);
    error = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	if (error != hipSuccess) Cleanup(false);
	error = hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);

    // Warm up
//    AddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread);
    call_gpu_function();
    error = hipGetLastError();
    if (error != hipSuccess) Cleanup(false);
    hipDeviceSynchronize();


    // Invoke kernel
//    AddVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread);
    call_gpu_function();
    error = hipGetLastError();
    if (error != hipSuccess) Cleanup(false);

	// Compute elapsed time 
    hipDeviceSynchronize();

    long long exec_time_nanoseconds = get_exec_time_in_nanoseconds(call_gpu_function); 
    
    //high_resolution_power_profile(call_gpu_function);


    double time = exec_time_nanoseconds/1e6; //in ms

    //high_resolution_power_profile(call_gpu_function);


	// Compute floating point operations per second.
    double nFlops = (double)N*(double)REPS ;
    double nFlopsPerSec = 1e3*nFlops/time;
    double nGFlopsPerSec = nFlopsPerSec*1e-9;
    //double nGFlopsPerSec = (1e3*N/(exec_time_nanoseconds/1e9))*1e-9;
    //double nGFlopsPerSec = 1e3*N/exec_time_nanoseconds;

	// Compute transfer rates.
    double nBytes = 3*4*(double)N*(double)REPS; // 2N words in, 1N word out
    double nBytesPerSec = 1e3*nBytes/time;
    double nGBytesPerSec = nBytesPerSec*1e-9;
    //double nGBytesPerSec = (1e3*nBytes/(exec_time_nanoseconds/1e9))*1e-9;
    //double nGBytesPerSec = 1e3*nBytes/exec_time_nanoseconds;

	// Report timing data.
    printf( "%d %d %d Time: %f (ms), GFlopsS: %f GBytesS: %f nytes: %f nBytesPerS: %f\n", GridWidth, BlockWidth, ValuesPerThread,
             time, nGFlopsPerSec, nGBytesPerSec, nBytes, nBytesPerSec);
     
    // Copy result from device memory to host memory
    error = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    if (error != hipSuccess) Cleanup(false);

    // Verify & report result
    for (i = 0; i < N; ++i) {
        float val = h_C[i];
        if (fabs(val - N) > 1e-5)
            break;
    }
    printf("Test %s \n", (i == N) ? "PASSED" : "FAILED");

	// Clean up and exit.
    Cleanup(true);
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
        
    // Free device vectors
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    error = hipDeviceReset();
    
    if (!noError || error != hipSuccess)
        printf("cuda malloc or cuda thread exit failed \n");
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
      exit(-1);
    }                         
}


