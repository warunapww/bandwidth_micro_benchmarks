#include "hip/hip_runtime.h"
#include "pp_dynamic_access_offchip_memory_char_vecadd_repeat.h"

__global__ void AddVectors(const char* A, const char* B, char* C, int N)
{
  int blockStartIndex  = blockIdx.x * blockDim.x * N;
  int threadStartIndex = blockStartIndex + threadIdx.x;
  int threadEndIndex   = threadStartIndex + N*blockDim.x;
  int i,t;

  for (t = 0; t < REPS; t++) {
    for( i=threadStartIndex; i<threadEndIndex; i=i+blockDim.x ){
        C[i] = A[i] + B[i];
    }
  }
}
